
#include <hip/hip_runtime.h>
#define INF 2000000000
#define GID(x, y) ( ((y)*N) + (x))

extern "C" {
__global__ void testOnGPU(int* capacityGPU, int* excessGPU, int* heightGPU, 
	int N, int s_x, int s_y, int t_x, int t_y, int* changed, int *mapGPU)
{
	// *changed = 0;
	// int blockId = blockIdx.x + blockIdx.y * gridDim.x; 
	//int u = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	
	int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int u = GID(x, y);
    // testing macro and coordinatce calculations
    // mapGPU[u] = x*100 + y;
    // mapGPU[u] = GID(x, y);
    // mapGPU[u] = ( u==GID(x, y) ? 1 : 0 );
    // return;



	// exit if source or sink
	if( (GID(s_x, s_y) == u) || (GID(t_x, t_y) == u)){
		return;
	}

	if(excessGPU[u] > 0){
		int e_temp = excessGPU[u];
		int h_temp = INF;
		int v_temp = -1;
		// int v_tempY = -1;
		int u_to_v = -1; 
		int v_to_u = -1;

		/* find lowest neighboor v_temp of u */

		// capacityGPU[4*ID+0] up
		// capacityGPU[4*ID+1] right
		// capacityGPU[4*ID+2] down
		// capacityGPU[4*ID+3] left

		// EDGE UP
		if ( capacityGPU[4 * u + 0] > 0 ){
			int v = GID(x, y-1);
			if (heightGPU[v] < h_temp) {
				u_to_v = 0;
				v_to_u = 2;
				v_temp = v;
				h_temp = heightGPU[v];
			}
		}
		// EDGE DOWN
		if ( capacityGPU[4 * u + 2] > 0 ){
			int v = GID(x, y+1);
			if (heightGPU[v] < h_temp) {
				u_to_v = 2;
				v_to_u = 0;
				v_temp = v;
				h_temp = heightGPU[v];
			}
		}
		//EDGE LEFT
		if ( capacityGPU[4 * u + 1] > 0 ){
			int v = GID(x+1, y);
			if (heightGPU[v] < h_temp) {
				u_to_v = 1;
				v_to_u = 3;
				v_temp = v;
				h_temp = heightGPU[v];
			}
		}
		// EDGE RIGHT
		if ( capacityGPU[4 * u + 3] > 0 ){
			int v = GID(x-1, y);
			if (heightGPU[v] < h_temp) {
				u_to_v = 3;
				v_to_u = 1;
				v_temp = v;
				h_temp = heightGPU[v];
			}
		}
		// //EDGE UP
		// if(y > 0){
		// 	int v = GID(x, y-1);
		// 	if(heightGPU[v] < h_temp && capacityGPU[4*u+0] > 0){
		// 		v_temp = GID(x, y-1);
		// 		// v_tempY = y-1;
		// 		u_to_v = 0;
		// 		v_to_u = 2;
		// 		h_temp = heightGPU[v];
		// 	}
		// }

		// //EDGE DOWN
		// if(y < N-1){
		// 	int v = GID(x, y+1);
		// 	if(heightGPU[v] < h_temp && capacityGPU[4*u+2] > 0){
		// 		v_temp = GID(x, y+1);
		// 		// v_tempY = y+1;
		// 		u_to_v = 2;
		// 		v_to_u = 0;
		// 		h_temp = heightGPU[v];
		// 	}
		// }

		// //EDGE LEFT
		// if(x > 0){
		// 	int v = GID(x-1, y);
		// 	if(heightGPU[v] < h_temp && capacityGPU[4*u+1] > 0){
		// 		v_temp = GID(x-1, y);
		// 		// v_tempY = y;
		// 		u_to_v = 1;
		// 		v_to_u = 3;
		// 		h_temp = heightGPU[v];
		// 	}
		// }

		// //EDGE RIGHT
		// if(x < N-1){
		// 	int v = GID(x+1, y);
		// 	if(heightGPU[v] < h_temp && capacityGPU[4*u+3] > 0){
		// 		v_temp = GID(x+1, y);
		// 		// v_tempY = y;
		// 		u_to_v = 3;
		// 		v_to_u = 1;
		// 		h_temp = heightGPU[v];
		// 	}
		// }

		if(v_temp == -1 ){
			printf("Something is wrong: %d %d\n", x, y);
			return;
		}

		if(heightGPU[u] > h_temp){
			int d = min(e_temp, capacityGPU[4*u + u_to_v]);
			atomicSub(&capacityGPU[4*u+u_to_v], d);			//edge from u to v_temp
			atomicAdd(&capacityGPU[4*v_temp + v_to_u], d);	//edge from v_temp to u
			
			atomicSub(&excessGPU[u], d);
			atomicAdd(&excessGPU[v_temp], d);

			
		} else {
			heightGPU[u] = h_temp + 1;
		}
		*changed = 1;
		//mapGPU[u] = capacityGPU[u];
	}
}

};
