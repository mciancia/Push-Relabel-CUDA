
#include <hip/hip_runtime.h>
#define INF 2000000000
#define GID(x, y) ( ((y)*N) + (x))

// while excess(u) > 0 do
// 	e_temp = excess(u)
// 	v_temp = null 
	

// 	h_temp = INF
	
// 	for each (u, v) ∈ Ef 
// 		if height(v) < h_temp [height(v_temp)], then
// 			v_temp = v
// 			h_temp = height(v)
// 	end for 
	
// 	if height(u) > h_temp, then  
// 		d = min(e_temp , cf(u, v_temp))
// 		cf(u,v_temp)−=d 
// 		cf(v_temp,u)+=d 
// 		excess(u)−=d
// 		excess(v_temp) += d
// 	else
// 		height(u) = h_temp + 1

extern "C" {
__global__ void testOnGPU(int* capacityGPU, int* excessGPU, int* heightGPU, 
	int N, int s_x, int s_y, int t_x, int t_y, int* changed, int *mapGPU)
{

	int blockId = blockIdx.x + blockIdx.y * gridDim.x; 
	int u = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	
	int x = threadIdx.x+blockIdx.x*blockDim.x;
    int y = threadIdx.y+blockIdx.y*blockDim.y;

    // testing macro and coordinatce calculations
    // mapGPU[u] = x*100 + y;
    // mapGPU[u] = GID(x, y);
    // mapGPU[u] = ( u==GID(x, y) ? 1 : 0 );
    // return;



	// exit if source or sink
	if( (GID(s_x, s_y) == u) || (GID(t_x, t_y) == u)){
		return;
	}

	if(excessGPU[u] > 0){
		int e_temp = excessGPU[u];
		int h_temp = INF;
		int v_tempX = -1;
		int v_tempY = -1;
		int u_to_v = -1, v_to_u = -1;

		/* find lowest neighboor v_temp of u */

		// capacityGPU[4*ID+0] up
		// capacityGPU[4*ID+1] right
		// capacityGPU[4*ID+2] down
		// capacityGPU[4*ID+3] left

		//EDGE UP
		if(y > 0){
			int v = GID(x, y-1);
			if(heightGPU[v] < h_temp){
				v_tempX = x;
				v_tempY = y-1;
				u_to_v = 0;
				v_to_u = 2;
				h_temp = heightGPU[v];
			}
		}

		//EDGE DOWN
		if(y < N-1){
			int v = GID(x, y+1);
			if(heightGPU[v] < h_temp){
				v_tempX = x;
				v_tempY = y+1;
				u_to_v = 2;
				v_to_u = 0;
				h_temp = heightGPU[v];
			}
		}

		//EDGE LEFT
		if(x > 0){
			int v = GID(x-1, y);
			if(heightGPU[v] < h_temp){
				v_tempX = x-1;
				v_tempY = y;
				u_to_v = 1;
				v_to_u = 3;
				h_temp = heightGPU[v];
			}
		}

		//EDGE RIGHT
		if(x < N-1){
			int v = GID(x+1, y);
			if(heightGPU[v] < h_temp){
				v_tempX = x+1;
				v_tempY = y;
				u_to_v = 3;
				v_to_u = 1;
				h_temp = heightGPU[v];
			}
		}

		if(v_tempX == -1 || v_tempY == -1 || u_to_v == -1 || v_to_u == -1){
			printf("Something is wrong\n");
			return;
		}

		if(heightGPU[u] > h_temp){
			int d = min(e_temp, capacityGPU[4*GID(x, y) + u_to_v]);
			atomicSub(&capacityGPU[4*GID(x, y)+u_to_v], d);					//edge from u to v_temp
			atomicAdd(&capacityGPU[4*GID(v_tempX, v_tempY) + v_to_u], d);	//edge from v_temp to u
			
			atomicSub(&excessGPU[u], d);
			atomicAdd(&excessGPU[GID(v_tempX, v_tempY)], d);

			*changed = 1;
		} else {
			atomicAdd(&heightGPU[u], (h_temp+1) );
		}
		mapGPU[u] = heightGPU[u];
	}
}

};
